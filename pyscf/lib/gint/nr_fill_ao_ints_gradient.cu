#include <cassert>

#include "nr_fill_ao_ints_gradient.cuh"
#include "g2e_gradient.cu"
#include "cuda_alloc.cuh"

#define print printf("The code has arrived here\n");

void GINTinit_uw_s1_nabla1i(double *uw_buf, BasisProdOffsets *offsets,
                            GINTEnvVars *envs, BasisProdCache *bpcache)
{
  size_t ntasks_ij = offsets->ntasks_ij;
  size_t ntasks_kl = offsets->ntasks_kl;
  int nprim_ij = envs->nprim_ij;
  int nprim_kl = envs->nprim_kl;
  int nroots = envs->nrys_roots + 1;
  int strides = envs->nprim_ij * envs->nprim_kl * nroots * 2;
  int n_primitive_pairs = bpcache->primitive_pairs_locs[bpcache->ncptype];
  double *a12 = bpcache->aexyz;
  double *x12 = bpcache->aexyz + n_primitive_pairs * 2;
  double *y12 = bpcache->aexyz + n_primitive_pairs * 3;
  double *z12 = bpcache->aexyz + n_primitive_pairs * 4;

#pragma omp parallel
  {
    int ij, kl, task_ij, task_kl, prim_ij, prim_kl;
    size_t n;
    double *uw;
#pragma omp for schedule(static)
    for (n = 0; n < ntasks_ij*ntasks_kl; n++) {
      task_ij = n % ntasks_ij;
      task_kl = n / ntasks_ij;
      prim_ij = offsets->primitive_ij + task_ij * nprim_ij;
      prim_kl = offsets->primitive_kl + task_kl * nprim_kl;
      uw = uw_buf + n * strides;
      for (ij = prim_ij; ij < prim_ij+nprim_ij; ij++) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; kl++) {
          double aij = a12[ij];
          double xij = x12[ij];
          double yij = y12[ij];
          double zij = z12[ij];
          double akl = a12[kl];
          double xkl = x12[kl];
          double ykl = y12[kl];
          double zkl = z12[kl];
          double rx = xij - xkl;
          double ry = yij - ykl;
          double rz = zij - zkl;
          double aijkl = aij + akl;
          double a0 = aij * akl / aijkl;
          double x = a0 * (rx * rx + ry * ry + rz * rz);
          double *u = uw;
          double *w = uw + nroots;
          CINTrys_roots(nroots, x, u, w);
          uw += nroots * 2;
        } }
    }
  }

__host__
static int GINTfill_nabla1i_int2e_tasks(ERITensor *eri,
                                        BasisProdOffsets *offsets,
                                        GINTEnvVars *envs)
{
  int nrys_roots = envs->nrys_roots;
  int ntasks_ij = offsets->ntasks_ij;
  int ntasks_kl = offsets->ntasks_kl;
  assert(ntasks_kl < 65536*THREADSY);
  int type_ijkl;

  dim3 threads(THREADSX, THREADSY);
  dim3 blocks((ntasks_ij+THREADSX-1)/THREADSX, (ntasks_kl+THREADSY-1)/THREADSY);
  switch (nrys_roots) {
    case 1:
      type_ijkl = (envs->i_l << 3) | (envs->j_l << 2) | (envs->k_l << 1) | envs->l_l;
      //GINTfill_int2e_kernel<1, GOUTSIZE1> <<<blocks, threads>>>(*offsets);
      switch (type_ijkl) {
        case 0b0000: GINTfill_nabla1i_int2e_kernel0000<<<blocks, threads>>>(*eri, *offsets); break;
        default:
          fprintf(stderr, "troots=1 ype_ijkl %d\n", type_ijkl);
      }
      break;

    default:
      fprintf(stderr, "rys roots %d\n", nrys_roots);
      return 1;
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error of GINTfill_int2e_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

extern "C" {__host__

int GINTfill_nabla1i_int2e(BasisProdCache *bpcache,
                           double *eri, int nao,
                           int *strides, int *ao_offsets,
                           int *bins_locs_ij, int *bins_locs_kl, int nbins,
                           int cp_ij_id, int cp_kl_id)
{
  ContractionProdType *cp_ij = bpcache->cptype + cp_ij_id;
  ContractionProdType *cp_kl = bpcache->cptype + cp_kl_id;
  GINTEnvVars envs;
  GINTinit_EnvVars(&envs, cp_ij, cp_kl);
  if (envs.nrys_roots >= 8) {
    return 2;
  }

  if (envs.nrys_roots > 2) {
    int16_t *idx4c = (int16_t *)malloc(sizeof(int16_t) * envs.nf * 3);
    int *idx_ij = (int *)malloc(sizeof(int) * envs.nfi * envs.nfj * 3);
    int *idx_kl = (int *)malloc(sizeof(int) * envs.nfk * envs.nfl * 3);
    GINTinit_2c_gidx(idx_ij, cp_ij->l_bra, cp_ij->l_ket);
    GINTinit_2c_gidx(idx_kl, cp_kl->l_bra, cp_kl->l_ket);
    GINTinit_4c_idx(idx4c, idx_ij, idx_kl, &envs);
    if (envs.nf > NFffff) {
      DEVICE_INIT(int16_t, d_idx4c, idx4c, envs.nf * 3);
      envs.idx = d_idx4c;
    } else {
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_idx4c), idx4c, sizeof(int16_t)*envs.nf*3));
    }
    free(idx4c);
    free(idx_ij);
    free(idx_kl);
  }

  int nrys_roots_gradient = envs.nrys_roots + 1;

  // Data and buffers to be allocated on-device. Allocate them here to
  // reduce the calls to malloc
  int nroots2 = nrys_roots_gradient * 2;
  int kl_bin, ij_bin1;
  double *uw_buf, *d_uw;
  size_t uw_size = 0;
  if (nrys_roots_gradient + 1 > POLYFIT_ORDER) {
    for (kl_bin = 0; kl_bin < nbins; ++kl_bin) {
      ij_bin1 = nbins - kl_bin;
      int bas_ij0 = bins_locs_ij[0];
      int bas_ij1 = bins_locs_ij[ij_bin1];
      int bas_kl0 = bins_locs_kl[kl_bin];
      int bas_kl1 = bins_locs_kl[kl_bin+1];
      int ntasks_ij = bas_ij1 - bas_ij0;
      int ntasks_kl = bas_kl1 - bas_kl0;
      uw_size = MAX(uw_size, ntasks_ij * ntasks_kl);
    }
    uw_size *= envs.nprim_ij * envs.nprim_kl * nroots2;
    checkCudaErrors(hipHostAlloc(&uw_buf, sizeof(double) * uw_size,
                                  hipHostMallocMapped));
    checkCudaErrors(hipMalloc(&d_uw, sizeof(double) * uw_size));
    envs.uw = d_uw;
  }
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_envs), &envs, sizeof(GINTEnvVars)));
  // move bpcache to constant memory
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_bpcache), bpcache, sizeof(BasisProdCache)));

  ERITensor eritensor;
  eritensor.stride_j = strides[1];
  eritensor.stride_k = strides[2];
  eritensor.stride_l = strides[3];
  eritensor.n_elem = nao * strides[3];
  eritensor.ao_offsets_k = ao_offsets[2];
  eritensor.ao_offsets_l = ao_offsets[3];
  eritensor.nao = nao;
  eritensor.data = eri;

  BasisProdOffsets offsets;
  int *bas_pairs_locs = bpcache->bas_pairs_locs;
  int *primitive_pairs_locs = bpcache->primitive_pairs_locs;
  for (kl_bin = 0; kl_bin < nbins; kl_bin++) {
    int bas_kl0 = bins_locs_kl[kl_bin];
    int bas_kl1 = bins_locs_kl[kl_bin+1];
    int ntasks_kl = bas_kl1 - bas_kl0;
    if (ntasks_kl <= 0) {
      continue;
    }
    // ij_bin + kl_bin < nbins <~> e_ij*e_kl < cutoff
    ij_bin1 = nbins - kl_bin;
    int bas_ij0 = bins_locs_ij[0];
    int bas_ij1 = bins_locs_ij[ij_bin1];
    int ntasks_ij = bas_ij1 - bas_ij0;
    if (ntasks_ij <= 0) {
      continue;
    }
    offsets.ntasks_ij = ntasks_ij;
    offsets.ntasks_kl = ntasks_kl;
    offsets.bas_ij = bas_pairs_locs[cp_ij_id] + bas_ij0;
    offsets.bas_kl = bas_pairs_locs[cp_kl_id] + bas_kl0;
    offsets.primitive_ij = primitive_pairs_locs[cp_ij_id] + bas_ij0 * envs.nprim_ij;
    offsets.primitive_kl = primitive_pairs_locs[cp_kl_id] + bas_kl0 * envs.nprim_kl;

    if (nrys_roots_gradient > POLYFIT_ORDER) {
      // move rys roots and weights to device
      GINTinit_uw_s1(uw_buf, &offsets, &envs, bpcache);
      uw_size = (size_t)ntasks_ij * ntasks_kl * envs.nprim_ij * envs.nprim_kl * nroots2;
      checkCudaErrors(hipMemcpy(d_uw, uw_buf, sizeof(double) * uw_size,
                                 hipMemcpyHostToDevice));
    }
    int err = GINTfill_nabla1i_int2e_tasks(&eritensor, &offsets, &envs);
    if (err != 0) {
      return err;
    }
  }

  if (nrys_roots_gradient > POLYFIT_ORDER) {
    checkCudaErrors(hipHostFree(uw_buf));
    FREE(d_uw);
  }
  if (envs.nf > NFffff) {
    FREE(envs.idx);
  }
  return 0;

}
}
